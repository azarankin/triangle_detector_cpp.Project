#include "hip/hip_runtime.h"
#include <shape_detector_common.h>
#include <utils_cuda_opencv.cuh>


#include <hip/hip_runtime.h>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/cudaimgproc.hpp> // cv::cuda::cvtColor
#include <opencv2/cudafilters.hpp>

std::string triangle_print_message() 
{
    std::cout << "\t\t\t\t\t\t\tPrinting from CUDA OpenCV Triangel Detector library!" << std::endl;
    return "cuda_opencv_triangle_detector";
}


std::vector<cv::Point> find_shape_contour(const std::string& address)
{
    // Load image
    cv::Mat host_img = cv::imread(address);
    if (host_img.empty())
        throw std::runtime_error("Failed to load image: " + address);
    
    // Upload to GPU
    cv::cuda::GpuMat gpu_img(host_img);

    // Convert to grayscale
    cv::cuda::GpuMat gpu_gray;
    cuda_gray_filter(gpu_img ,gpu_gray); // , "find_shape_contour__gray"

    // Gaussian blur
    cv::cuda::GpuMat gpu_blur;
    cuda_gaussian_blur_filter(gpu_gray, gpu_blur); // , "find_shape_contour__blur"

    // Threshold
    cv::cuda::GpuMat gpu_thresh;
    cv::Mat thresh1;
    cuda_threshold_filter(gpu_blur, gpu_thresh); // , "find_shape_contour__thresh"

    gpu_thresh.download(thresh1);

    // Find contours (CPU only)
    std::vector<std::vector<cv::Point>> contours_template;
    find_contours(contours_template, thresh1);


    // Sort contours by area
    auto second_largest_contour = get_second_largest_contour(contours_template);

    // Draw the second largest contour
    return second_largest_contour;
}


std::vector<std::vector<cv::Point>> contour_compare(
    cv::Mat& target_frame, 
    const std::vector<cv::Point>& template_contour,
    double match_threshold, 
    double min_area_ratio, 
    double max_area_ratio)
{
    // Upload image to GPU
    cv::cuda::GpuMat gpu_frame(target_frame);

    // Gray filter (CUDA)
    cv::cuda::GpuMat gpu_gray;
    cuda_gray_filter(gpu_frame ,gpu_gray); // , "contour_compare__gray"

    // Gaussian blur (CUDA)
    cv::cuda::GpuMat gpu_blur;
    cuda_gaussian_blur_filter(gpu_gray, gpu_blur); // , "contour_compare__blur"

    // Adaptive threshold (CUDA)
    cv::cuda::GpuMat gpu_thresh;
    cuda_adaptive_threshold_filter(gpu_blur, gpu_thresh); // , "contour_compare__thresh"

    // Download the thresholded image back to CPU
    cv::Mat thresh2;
    gpu_thresh.download(thresh2);

    // Contours (CPU)
    std::vector<std::vector<cv::Point>> contours_target;
    find_contours(contours_target, thresh2);

    std::vector<std::vector<cv::Point>> closest_contours = find_closest_contours(target_frame, template_contour, contours_target, match_threshold, min_area_ratio, max_area_ratio);

    return closest_contours;
}


void create_triangle_image(const std::string& filename, cv::Size size, int margin) 
{
    int width = size.width;
    int height = size.height;

    uchar3* d_img;
    size_t imgSize = width * height * sizeof(uchar3);
    hipMalloc(&d_img, imgSize);
    
    dim3 block(256);  // כל בלוק ממלא שורות (thread לכל y)
    dim3 grid((height + block.x - 1) / block.x);
    drawEquilateralTriangleKernel<<<grid, block>>>(d_img, width, height, margin);

    hipDeviceSynchronize();

    // העברה חזרה ל־CPU וכתיבה
    std::vector<uchar3> h_img(width * height);
    hipMemcpy(h_img.data(), d_img, imgSize, hipMemcpyDeviceToHost);
    hipFree(d_img);

    // עטיפה לתוך cv::Mat
    cv::Mat img(height, width, CV_8UC3, h_img.data());
    cv::imwrite(filename, img);

    std::cout << "Triangle image (CUDA) saved as " << filename << std::endl;
}


